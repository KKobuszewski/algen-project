#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <omp.h>

#include "Eigensolver.hpp"
#include "cuerrors.h"



Eigensolver::Eigensolver(unsigned matrix_size, unsigned number_matrices, unsigned number_streams):
m(matrix_size),lda(matrix_size), _number_matrices(number_matrices), _number_streams(number_streams)
{
    //hipError_t cudaStat = hipSuccess;
    d_A = NULL;
    d_B = NULL;
    d_W = NULL;
    d_work = NULL;
    devInfo = NULL;
    
    printf("Initializing cusolver for matrices %dx%d and %u streams\n",m,lda,number_streams);
    
    
    // allocate memory for matrices
    CUDA_CALL(  hipMalloc( (void**) &d_A, sizeof(hipDoubleComplex) *lda*m * _number_streams )  );
    CUDA_CALL(  hipMalloc( (void**) &d_B, sizeof(hipDoubleComplex) *lda*m * _number_streams )  );
    CUDA_CALL(  hipMalloc( (void**) &d_W, sizeof(double) * m * _number_streams )              );
    CUDA_CALL(  hipMalloc( (void**) &devInfo, sizeof(int) * _number_matrices )                );
    printf("Memory allocated.\n");
    
    // create multiple cuda streams and create handles
    streams = (hipStream_t*) malloc( _number_streams * sizeof(hipStream_t) );
    cusolvers = (hipsolverHandle_t*) malloc( _number_streams * sizeof(hipsolverHandle_t) );
    for (unsigned ii=0; ii < _number_streams; ii++)
    {
        hipStreamCreate(&streams[ii]);
        cusolvers[ii] = NULL;
        
        assert( HIPSOLVER_STATUS_SUCCESS == hipsolverDnCreate(&cusolvers[ii]) );
        assert( HIPSOLVER_STATUS_SUCCESS == hipsolverSetStream(cusolvers[ii],streams[ii]) );
    }
    printf("Initializing streams..\n");
    
    // step 3: query working space of sygvd
    const hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    const hipsolverEigMode_t jobz  = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    const hipblasFillMode_t  uplo  = HIPBLAS_FILL_MODE_LOWER;
    lwork = 0;
    CUSOLVER_CALL(  hipsolverDnZhegvd_bufferSize(cusolvers[0], itype, jobz, uplo, m, d_A, lda, d_B, lda, d_W, &lwork)  );
    CUDA_CALL(  hipMalloc( (void**) &d_work, sizeof(hipDoubleComplex) * lwork * _number_streams )  );
    
    printf("Cusolver prepared.\n");
    printf("lwork: %d\n",lwork);
    printf("\n");
}

Eigensolver::~Eigensolver()
{
    // destroy cusolverfor (unsigned ii=0; ii < _number_streams; ii++)
    for (unsigned ii=0; ii < _number_streams; ii++)
    {
        assert( HIPSOLVER_STATUS_SUCCESS == hipsolverDnDestroy(cusolvers[ii]) );
    }
    
    // deallocate memory
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cusolvers) free(cusolvers);
    if (streams) free(streams);
}

/*
 * Solves generalized eigenvalue problem for many matrices A_i v_i = \lambda_i B_i v_i.
 * 
 * A - hamiltonians
 * B - overlaps
 * V - 
 * W - 
 * 
 */
void Eigensolver::find_generalized_eigvals_batched(
    std::complex<double> **A, std::complex<double> **B,
    std::complex<double> **V, double **E)
{
    const unsigned batch_size = lda*m;
    const hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1;       // A*x = (lambda)*B*x
    const hipsolverEigMode_t jobz  = HIPSOLVER_EIG_MODE_VECTOR;  // compute eigenvalues and eigenvectors.
    const hipblasFillMode_t  uplo  = HIPBLAS_FILL_MODE_LOWER;
    
    if (_number_matrices == 1)
    {
        fprintf(stderr,"Case of single dense matrix not implemented!\n");
        exit(EXIT_FAILURE);
    }
    else
    {
        #pragma omp parallel for num_threads(_number_matrices)
        for (unsigned ii=0; ii < _number_matrices; ii++)
        {
            int info_gpu = 0;
            //hipError_t cudaStat = hipSuccess;
            //hipsolverStatus_t cusolver_status;
            
            CUDA_CALL(  hipMemcpy(d_A + batch_size*ii, (hipDoubleComplex*) A[ii],
                                  sizeof(hipDoubleComplex) * batch_size, hipMemcpyHostToDevice)  );
            CUDA_CALL(  hipMemcpy(d_B + batch_size*ii, (hipDoubleComplex*) B[ii],
                                  sizeof(hipDoubleComplex) * batch_size, hipMemcpyHostToDevice)  );
            
            // step 4: compute spectrum of (A,B)
            CUSOLVER_CALL(  hipsolverDnZhegvd(cusolvers[ii], itype, jobz, uplo,
                            m, d_A + batch_size*ii, lda, d_B + batch_size*ii, lda, d_W + m*ii, d_work + lwork*ii, lwork, &devInfo[ii])  );
            CUDA_CALL(  hipDeviceSynchronize()  );
            
            //check the result
            CUDA_CALL(  hipMemcpy(E[ii], d_W + m*ii, sizeof(double)*m, hipMemcpyDeviceToHost)  );
            CUDA_CALL(  hipMemcpy(V[ii], d_A + batch_size*ii, sizeof(hipDoubleComplex)*lda*m, hipMemcpyDeviceToHost)  );
            CUDA_CALL(  hipMemcpy(&info_gpu, &devInfo[ii], sizeof(int), hipMemcpyDeviceToHost)  );
            
            if (info_gpu != 0)
                printf("%u. after hegvd: info_gpu = %d\n", ii, info_gpu);
            assert(0 == info_gpu);
        }
        
//         cusolverSpZcsreigs[Host](cusolverH, m, nnz, hipsparseMatDescr_t descrA,
//         const hipDoubleComplex *csrValA,
//         const int *csrRowPtrA,
//         const int *csrColIndA,
//         hipDoubleComplex left_bottom_corner,
//         hipDoubleComplex right_upper_corner,
//         int *num_eigs);
    }
}